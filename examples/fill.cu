#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>

#define CHUNK0_SIZE (64L * 1024L * 1024L * 1024L * 1024L + 0x55554000000L)
#define CHUNK1_SIZE (41L * 1024L * 1024L * 1024L * 1024L + 0x0ffc8000000L)
#define STRIDE_SIZE (1L * 1024L * 1024L)

#define BASE_ADDR   0x700000000000
#define DUMMY_ADDR  0x7F0000000000

#define PAGE_NUM    4000
//#define PAGE_NUM    2000
#define WAIT_TIME   10000000000L // about 5 seconds on RTX3080; TODO: adapt to A100

// Add error checking macro
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(EXIT_FAILURE);                                             \
 }                                                                 \
}

__global__ void 
loop(volatile uint64_t *page, uint64_t x)
{
  uint64_t y = x;
  volatile uint64_t *ptr; //volatile ensures the compiler doesn't optimize out memory accesses
  uint64_t clk0;
  uint64_t clk1;

  // print the addresses
  // printf("page[0]: %p, page: %p\n", (void *)page[0], (void *)page);
  
  while (y == x) {
    // init ptr as the first element of the page
    // iterates through a linked list of 64-bit unsigned integers
    // page[0]: start of the list
    // ptr != page checks if ptr has reached the list's end
    // ptr[2] accesses the third uint64_t in the node (offset 2), incrementing it.
    for (ptr = (uint64_t *)page[0]; ptr != page; ptr = (uint64_t *)ptr[0])
      ++ptr[2];
    
    clk0 = clock64();
    clk1 = 0;
    while (clk1 < WAIT_TIME)
      clk1 = clock64() - clk0;
    
    y = ptr[1];
  }
}

// Put values into memory
__global__ void
put(uint64_t *page, uint64_t x1, uint64_t x2)
{
  page[0] = x1;
  page[1] = x2;
}

int 
main(int argc, char *argv[])
{
  uint8_t *chunk0 = NULL;
  uint8_t *chunk1 = NULL;
  uint8_t *base = NULL;
  uint64_t *list[PAGE_NUM];
  uint64_t *dummy = NULL;
  
  hipDeviceReset();
  
  // hoard a large address space
  hipMallocManaged(&chunk0, CHUNK0_SIZE);
  hipMallocManaged(&chunk1, CHUNK1_SIZE);
  
  base = (uint8_t *)BASE_ADDR;
  for (int i = 0; i < PAGE_NUM; ++i)
    list[i] = (uint64_t *)(base + i * STRIDE_SIZE);
  dummy = (uint64_t *)DUMMY_ADDR;
  
  for (int i = 0; i < PAGE_NUM; ++i) {
    put<<<1, 1>>>(list[i], (uint64_t)list[(i + 1) % PAGE_NUM], 0xdeadbeef);
    cudaCheckError();
  }
  put<<<1, 1>>>(dummy, 0, 0);
  // cudaCheckError();
  hipDeviceSynchronize();
  // cudaCheckError();
  
  loop<<<1, 1>>>(list[0], 0xdeadbeef);
  // cudaCheckError();
  hipDeviceSynchronize();
  // cudaCheckError();
  
  hipFree(chunk0);
  hipFree(chunk1);
}



#include <hip/hip_runtime.h>
#include <stdio.h>

// Error checking macro
#define CUDA_CHECK(err) do { \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

// Matrix multiplication kernel
__global__ void matrixMul(const float *A, const float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if (row < N && col < N) {
        for (int i = 0; i < N; i++) {
            sum += A[row * N + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    const int N = 4; // Matrix size (NxN)
    const int size = N * N * sizeof(float);

    // Host matrices
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f; // A = all 1s
        h_B[i] = 2.0f; // B = all 2s
    }

    // Device matrices
    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, size));
    CUDA_CHECK(hipMalloc(&d_B, size));
    CUDA_CHECK(hipMalloc(&d_C, size));

    // Copy inputs to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    // Launch kernel
    dim3 threads(2, 2); // 2x2 threads per block
    dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);
    matrixMul<<<blocks, threads>>>(d_A, d_B, d_C, N);
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back
    CUDA_CHECK(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    // Verify result (C[i][j] should be N * 1.0 * 2.0 = 8.0 for 4x4)
    int correct = 1;
    for (int i = 0; i < N * N; i++) {
        if (h_C[i] != (float)(N * 2.0)) {
            correct = 0;
            break;
        }
    }
    printf("Matrix multiplication %s\n", correct ? "successful" : "failed");

    // Free memory
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}